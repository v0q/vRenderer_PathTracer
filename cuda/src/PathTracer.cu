#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <thrust/random/uniform_real_distribution.h>
#include <thrust/device_vector.h>

#include "PathTracer.cuh"
#include "RayIntersection.cuh"
#include "MathHelpers.cuh"

__constant__ __device__ uint numMeshes = 0;
__constant__ __device__ float kInvGamma = 1.f/2.2f;
__constant__ __device__ uint kSamps = 8;
__constant__ __device__ float kInvSamps = 1.f/8.f;

typedef struct Sphere {
	float m_r;       // radius
	float4 m_pos;
	float4 m_emission;
	float4 m_col;

	__device__ float intersect(const Ray *_r) const
	{ // returns distance, 0 if nohit
		float4 op = m_pos - _r->m_origin; // Solve t^2*d.d + 2*t*(o-p).d + (o-p).(o-p)-R^2 = 0
		float t;
		float eps = 1e-4;
		float b = dot(op, _r->m_dir);
		float det = b*b - dot(op, op) + m_r*m_r;
		if(det < 0)
			return 0;
		else
			det = sqrtf(det);
		return (t = b-det) > eps ? t : ((t = b+det) > eps ? t : 0.0);
	}
} Sphere;

__constant__ Sphere spheres[] = {			//Scene: radius, position, emission, color, material
	{ 1e5f, { 1e5f + 1.0f, 40.8f, 81.6f, 0.0f },			{ 0.075f, 0.f, 0.f, 0.0f }, { 0.75f, 0.0f, 0.0f, 0.0f } }, //Left
	{ 1e5f, { -1e5f + 99.0f, 40.8f, 81.6f, 0.0f },		{ 0.f, 0.075f, 0.f, 0.0f }, { 0.0f, 0.75f, 0.0f, 0.0f } }, //Right
	{ 1e5f, { 50.0f, 40.8f, 1e5f, 0.0f },							{ 0.0f, 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f, 0.0f } }, //Back
	{ 1e5f, { 50.0f, 40.8f, -1e5f + 600.0f, 0.0f },		{ 0.0f, 0.0f, 0.0f, 0.0f }, { 1.00f, 1.00f, 1.00f, 0.0f } }, //Frnt
	{ 1e5f, { 50.0f, 1e5f, 81.6f, 0.0f },							{ 0.0f, 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f, 0.0f } }, //Botm
	{ 1e5f, { 50.0f, -1e5f + 81.6f, 81.6f, 0.0f },		{ 0.0f, 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f, 0.0f } }, //Top
	{ 16.5f, { 27.0f, 16.5f, 47.0f, 0.0f },						{ 0.0f, 0.0f, 0.0f, 0.0f }, { 1.0f, 1.0f, 1.0f, 0.0f } }, // small sphere 1
	{ 16.5f, { 73.0f, 16.5f, 78.0f, 0.0f },						{ 0.0f, 0.0f, 0.0f, 0.0f }, { 1.0f, 1.0f, 1.0f, 0.0f } }, // small sphere 2
	{ 600.0f, { 50.0f, 681.6f - .77f, 81.6f, 0.0f },	{ 2.0f, 1.8f, 1.6f, 0.0f }, { 0.0f, 0.0f, 0.0f, 0.0f } }  // Light
};

__device__ inline bool intersectScene(const Ray *_ray, const vMesh *_scene, vHitData *_hitData)
{
	/* initialise t to a very large number,
	so t will be guaranteed to be smaller
	when a hit with the scene occurs */

	int n = sizeof(spheres)/sizeof(Sphere);
	float inf = 1e20f;
	float t = inf;

	/* check if the ray intersects each sphere in the scene */
	for(int i = 0; i < n; i++)  {
		/* float hitdistance = intersectSphere(&spheres[i], ray); */
		Sphere sphere = spheres[i]; /* create local copy of sphere */
		float dist = sphere.intersect(_ray);
		/* keep track of the closest intersection and hitobject found so far */
		if(dist != 0.0f && dist < t) {
			t = dist;
			_hitData->m_hitPoint = _ray->m_origin + _ray->m_dir * t;
			_hitData->m_normal = normalize(_hitData->m_hitPoint - sphere.m_pos);
			_hitData->m_color = sphere.m_col;
			_hitData->m_emission = sphere.m_emission;
		}
	}
	for(unsigned int j = 0; j < numMeshes; ++j)
	{
		if(intersectBVH(_scene[j].m_bvh, _ray))
		{
			for(unsigned int i = 0; i < _scene[j].m_triCount; ++i)
			{
				float dist = intersectTriangle(_scene[j].m_mesh[i].m_v1.m_vert, _scene[j].m_mesh[i].m_v2.m_vert, _scene[j].m_mesh[i].m_v3.m_vert, _ray);
				if(dist != 0.0f && dist < t) {
					t = dist;
					_hitData->m_hitPoint = _ray->m_origin + _ray->m_dir * t;
					_hitData->m_normal = _scene[j].m_mesh[i].m_v1.m_normal;
					_hitData->m_color = make_float4(1.f, 1.f, 1.f, 0.f);
					_hitData->m_emission = make_float4(0.f, 0.f, 4.5f, 0.f);
				}
			}
		}
	}

	return t < inf; /* true when ray interesects the scene */
}

__device__ static unsigned int hash(unsigned int *seed0, unsigned int *seed1)
{
 *seed0 = 36969 * ((*seed0) & 65535) + ((*seed0) >> 16); // hash the seeds using bitwise AND and bitshifts
 *seed1 = 18000 * ((*seed1) & 65535) + ((*seed1) >> 16);

	return *seed0**seed1;
}

__device__ float4 trace(const Ray *_camray, const vMesh *_scene, unsigned int *_seed0, unsigned int *_seed1)
{
	Ray ray = *_camray;

	float4 accum_color = make_float4(0.0f, 0.0f, 0.0f, 0.f);
	float4 mask = make_float4(1.0f, 1.0f, 1.0f, 0.f);

	for(unsigned int bounces = 0; bounces < 4; bounces++)
	{
		vHitData hitData;

		if(!intersectScene(&ray, _scene, &hitData)) {
			return make_float4(0.f, 0.f, 0.f, 0.f);
		}

		unsigned int seed = hash(_seed0, _seed1);
		thrust::default_random_engine rng(seed);
		thrust::random::uniform_real_distribution<float> uniformDist(0, 1);

		/* compute the surface normal and flip it if necessary to face the incoming ray */
		float4 normal_facing = dot(hitData.m_normal, ray.m_dir) < 0.0f ? hitData.m_normal : hitData.m_normal * (-1.0f);
		/* compute two random numbers to pick a random point on the hemisphere above the hitpoint*/
		float rand1 = 2.0f * PI * uniformDist(rng);
		float rand2 = uniformDist(rng);
		float rand2s = sqrt(rand2);

		/* create a local orthogonal coordinate frame centered at the hitpoint */
		float4 w = normal_facing;
		float4 axis = fabs(w.x) > 0.1f ? make_float4(0.0f, 1.0f, 0.0f, 0.f) : make_float4(1.0f, 0.0f, 0.0f, 0.f);
		float4 u = normalize(cross(axis, w));
		float4 v = cross(w, u);

		/* use the coordinte frame and random numbers to compute the next ray direction */
		float4 newdir = normalize(u * cos(rand1)*rand2s + v*sin(rand1)*rand2s + w*sqrt(1.0f - rand2));

		/* add a very small offset to the hitpoint to prevent self intersection */
		ray.m_origin = hitData.m_hitPoint + normal_facing * 0.05f;
		ray.m_dir = newdir;

		/* add the colour and light contributions to the accumulated colour */
		accum_color += mask * hitData.m_emission;

		/* the mask colour picks up surface colours at each bounce */
		mask *= hitData.m_color;

		/* perform cosine-weighted importance sampling for diffuse surfaces*/
		mask *= dot(newdir, normal_facing);
		mask *= 2;
	}

	return accum_color;
}

__global__ void render(hipSurfaceObject_t _tex, const vMesh *_scene, float4 *_colors, float4 *_cam, float4 *_dir, unsigned int _w, unsigned int _h, unsigned int _frame, unsigned int _time)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(x < _w && y < _h) {

		unsigned int ind = x + y*_w;
    unsigned int s1 = x * _frame;
		unsigned int s2 = y * _time;

    if(_frame == 1) {
			_colors[ind] = make_float4(0.f, 0.f, 0.f, 0.f);
    }

		Ray camera(*_cam, *_dir);

		float4 cx = make_float4(_w * .5135 / _h, 0.0f, 0.0f, 0.0f); // ray direction offset in x direction
		float4 cy = normalize(cross(cx, camera.m_dir)) * .5135; // ray direction offset in y direction (.5135 is field of view angle)

		for(unsigned int s = 0; s < kSamps; s++) {  // samples per pixel
			// compute primary ray direction
			float4 d = camera.m_dir + cx*((.25 + x) / _w - .5) + cy*((.25 + y) / _h - .5);
			// create primary ray, add incoming radiance to pixelcolor
			Ray newcam(camera.m_origin + d * 40, normalize(d));
			_colors[ind] += trace(&newcam, _scene, &s1, &s2) * (kInvSamps);
		}

		float coef = 1.f/_frame;
		unsigned char r = (unsigned char)(powf(clamp(_colors[ind].x*coef, 0.0, 1.0), kInvGamma) * 255);
		unsigned char g = (unsigned char)(powf(clamp(_colors[ind].y*coef, 0.0, 1.0), kInvGamma) * 255);
		unsigned char b = (unsigned char)(powf(clamp(_colors[ind].z*coef, 0.0, 1.0), kInvGamma) * 255);

		uchar4 data = make_uchar4(r, g, b, 0xff);
		surf2Dwrite(data, _tex, x*sizeof(uchar4), y);
	}
}

void cu_runRenderKernel(hipSurfaceObject_t _texture, const vMesh *_scene, float4 *_colorArr, float4 *_cam, float4 *_dir, unsigned int _w, unsigned int _h, unsigned int _frame, unsigned int _time)
{
	dim3 dimBlock(16, 16);
	dim3 dimGrid((_w / dimBlock.x),
							 (_h / dimBlock.y));

	render<<<dimGrid, dimBlock>>>(_texture, _scene, _colorArr, _cam, _dir, _w, _h, _frame, _time);
}

void cu_updateMeshCount(unsigned int _numMeshes)
{
	hipMemcpyToSymbol(HIP_SYMBOL(numMeshes), &_numMeshes, sizeof(unsigned int));
}

void cu_fillFloat4(float4 *d_ptr, float4 _val, unsigned int _size)
{
	thrust::device_ptr<float4> ptr = thrust::device_pointer_cast(d_ptr);
	thrust::fill(ptr, ptr + _size, _val);
}
