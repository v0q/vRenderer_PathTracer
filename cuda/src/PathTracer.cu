#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <thrust/random/uniform_real_distribution.h>
#include <thrust/device_vector.h>

#include "PathTracer.cuh"
#include "RayIntersection.cuh"
#include "MathHelpers.cuh"

#define invGamma 1.f/2.2f

__constant__ Sphere spheres[] = {			//Scene: radius, position, emission, color, material
	{ 1e5f, { 1e5f + 1.0f, 40.8f, 81.6f, 0.0f },			{ 0.075f, 0.f, 0.f, 0.0f }, { 0.75f, 0.0f, 0.0f, 0.0f } }, //Left
	{ 1e5f, { -1e5f + 99.0f, 40.8f, 81.6f, 0.0f },		{ 0.f, 0.075f, 0.f, 0.0f }, { 0.0f, 0.75f, 0.0f, 0.0f } }, //Right
	{ 1e5f, { 50.0f, 40.8f, 1e5f, 0.0f },							{ 0.0f, 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f, 0.0f } }, //Back
	{ 1e5f, { 50.0f, 40.8f, -1e5f + 600.0f, 0.0f },		{ 0.0f, 0.0f, 0.0f, 0.0f }, { 1.00f, 1.00f, 1.00f, 0.0f } }, //Frnt
	{ 1e5f, { 50.0f, 1e5f, 81.6f, 0.0f },							{ 0.0f, 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f, 0.0f } }, //Botm
	{ 1e5f, { 50.0f, -1e5f + 81.6f, 81.6f, 0.0f },		{ 0.0f, 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f, 0.0f } }, //Top
//	{ 16.5f, { 27.0f, 16.5f, 47.0f, 0.0f },						{ 0.0f, 0.0f, 0.0f, 0.0f }, { 1.0f, 1.0f, 1.0f, 0.0f } }, // small sphere 1
//	{ 16.5f, { 73.0f, 16.5f, 78.0f, 0.0f },						{ 0.0f, 0.0f, 0.0f, 0.0f }, { 1.0f, 1.0f, 1.0f, 0.0f } }, // small sphere 2
	{ 600.0f, { 50.0f, 681.6f - .77f, 81.6f, 0.0f },	{ 2.0f, 1.8f, 1.6f, 0.0f }, { 0.0f, 0.0f, 0.0f, 0.0f } }  // Light
};

__device__ inline bool intersectScene(const Ray *_ray,  const vMesh *_scene, vHitData *_hitData)
{
	/* initialise t to a very large number,
	so t will be guaranteed to be smaller
	when a hit with the scene occurs */

	int n = sizeof(spheres)/sizeof(Sphere);
	float inf = 1e20f;
	float t = inf;

	/* check if the ray intersects each sphere in the scene */
	for(int i = 0; i < n; i++)  {
		/* float hitdistance = intersectSphere(&spheres[i], ray); */
		Sphere sphere = spheres[i]; /* create local copy of sphere */
		float dist = sphere.intersect(_ray);
		/* keep track of the closest intersection and hitobject found so far */
		if(dist != 0.0f && dist < t) {
			t = dist;
			_hitData->m_hitPoint = _ray->m_origin + _ray->m_dir * t;
			_hitData->m_normal = normalize(_hitData->m_hitPoint - sphere.m_pos);
			_hitData->m_color = sphere.m_col;
			_hitData->m_emission = sphere.m_emission;
		}
	}
	if(intersectBoundingBox(_scene[0].m_bb, _ray))
	{
		for(unsigned int i = 0; i < _scene[0].m_triCount; ++i)
		{
			float dist = intersectTriangle(_scene[0].m_mesh[i].m_v1.m_vert, _scene[0].m_mesh[i].m_v2.m_vert, _scene[0].m_mesh[i].m_v3.m_vert, _ray);
			if(dist != 0.0f && dist < t) {
				t = dist;
				_hitData->m_hitPoint = _ray->m_origin + _ray->m_dir * t;
				_hitData->m_normal = _scene[0].m_mesh[i].m_v1.m_normal;
				_hitData->m_color = make_float4(1.f, 1.f, 1.f, 0.f);
				_hitData->m_emission = make_float4(0.f, 0.0f, 0.0f, 0.f);
			}
		}
	}

	return t < inf; /* true when ray interesects the scene */
}

__device__ static unsigned int hash(unsigned int *seed0, unsigned int *seed1)
{
 *seed0 = 36969 * ((*seed0) & 65535) + ((*seed0) >> 16); // hash the seeds using bitwise AND and bitshifts
 *seed1 = 18000 * ((*seed1) & 65535) + ((*seed1) >> 16);

	return *seed0**seed1;
}

__device__ float4 trace(const Ray *_camray, const vMesh *_scene, unsigned int *_seed0, unsigned int *_seed1)
{
	Ray ray = *_camray;

	float4 accum_color = make_float4(0.0f, 0.0f, 0.0f, 0.f);
	float4 mask = make_float4(1.0f, 1.0f, 1.0f, 0.f);

	for(unsigned int bounces = 0; bounces < 4; bounces++)
	{
		vHitData hitData;

		if(!intersectScene(&ray, _scene, &hitData)) {
			return make_float4(0.f, 0.f, 0.f, 0.f);
		}

		unsigned int seed = hash(_seed0, _seed1);
		thrust::default_random_engine rng(seed);
		thrust::random::uniform_real_distribution<float> uniformDist(0, 1);

		/* compute the surface normal and flip it if necessary to face the incoming ray */
		float4 normal_facing = dot(hitData.m_normal, ray.m_dir) < 0.0f ? hitData.m_normal : hitData.m_normal * (-1.0f);
		/* compute two random numbers to pick a random point on the hemisphere above the hitpoint*/
		float rand1 = 2.0f * PI * uniformDist(rng);
		float rand2 = uniformDist(rng);
		float rand2s = sqrt(rand2);

		/* create a local orthogonal coordinate frame centered at the hitpoint */
		float4 w = normal_facing;
		float4 axis = fabs(w.x) > 0.1f ? make_float4(0.0f, 1.0f, 0.0f, 0.f) : make_float4(1.0f, 0.0f, 0.0f, 0.f);
		float4 u = normalize(cross(axis, w));
		float4 v = cross(w, u);

		/* use the coordinte frame and random numbers to compute the next ray direction */
		float4 newdir = normalize(u * cos(rand1)*rand2s + v*sin(rand1)*rand2s + w*sqrt(1.0f - rand2));

		/* add a very small offset to the hitpoint to prevent self intersection */
		ray.m_origin = hitData.m_hitPoint + normal_facing * 0.05f;
		ray.m_dir = newdir;

		/* add the colour and light contributions to the accumulated colour */
		accum_color += mask * hitData.m_emission;

		/* the mask colour picks up surface colours at each bounce */
		mask *= hitData.m_color;

		/* perform cosine-weighted importance sampling for diffuse surfaces*/
		mask *= dot(newdir, normal_facing);
		mask *= 2;
	}

	return accum_color;
}

__global__ void render(hipSurfaceObject_t _tex, const vMesh *_scene, float4 *_colors, float4 *_cam, float4 *_dir, unsigned int _w, unsigned int _h, unsigned int _frame, unsigned int _time)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(x < _w && y < _h) {

		unsigned int ind = x + y*_w;
    unsigned int s1 = x * _frame;
		unsigned int s2 = y * _time;

    if(_frame == 1) {
			_colors[ind] = make_float4(0.f, 0.f, 0.f, 0.f);
    }

		Ray camera(*_cam, *_dir);

		float4 cx = make_float4(_w * .5135 / _h, 0.0f, 0.0f, 0.0f); // ray direction offset in x direction
		float4 cy = normalize(cross(cx, camera.m_dir)) * .5135; // ray direction offset in y direction (.5135 is field of view angle)

		unsigned int samps = 8;
		for(unsigned int s = 0; s < samps; s++) {  // samples per pixel
			// compute primary ray direction
			float4 d = camera.m_dir + cx*((.25 + x) / _w - .5) + cy*((.25 + y) / _h - .5);
			// create primary ray, add incoming radiance to pixelcolor
			Ray newcam(camera.m_origin + d * 40, normalize(d));
			_colors[ind] += trace(&newcam, _scene, &s1, &s2);
		}

		float coef = 1.f/(samps*_frame);
		unsigned char r = (unsigned char)(powf(clamp(_colors[ind].x*coef, 0.0, 1.0), invGamma) * 255);
		unsigned char g = (unsigned char)(powf(clamp(_colors[ind].y*coef, 0.0, 1.0), invGamma) * 255);
		unsigned char b = (unsigned char)(powf(clamp(_colors[ind].z*coef, 0.0, 1.0), invGamma) * 255);

		uchar4 data = make_uchar4(r, g, b, 0xff);
		surf2Dwrite(data, _tex, x*sizeof(uchar4), y);
	}
}

void cu_runRenderKernel(hipSurfaceObject_t _texture, const vMesh *_scene, float4 *_colorArr, float4 *_cam, float4 *_dir, unsigned int _w, unsigned int _h, unsigned int _frame, unsigned int _time)
{
	dim3 dimBlock(16, 16);
	dim3 dimGrid((_w / dimBlock.x),
							 (_h / dimBlock.y));

	render<<<dimGrid, dimBlock>>>(_texture, _scene, _colorArr, _cam, _dir, _w, _h, _frame, _time);
}

void cu_fillFloat4(float4 *d_ptr, float4 _val, unsigned int _size)
{
	thrust::device_ptr<float4> ptr = thrust::device_pointer_cast(d_ptr);
	thrust::fill(ptr, ptr + _size, _val);
}
